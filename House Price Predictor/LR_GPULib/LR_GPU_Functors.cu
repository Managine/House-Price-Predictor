#include "hip/hip_runtime.h"
#include <thrust\functional.h>

//
//Calculates the mean for a column major ordered matrix
//
struct MeanFunctor : public thrust::unary_function<int, float>
{
	float * trainingData;
	unsigned int trainingDataCount;
	unsigned int featureCount;

	MeanFunctor(float * _trainingData, unsigned int _trainingDataCount, unsigned int _featureCount) : trainingData(_trainingData), trainingDataCount(_trainingDataCount), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid)
  {
	  float sum = 0;
	  for(int i = 0; i < trainingDataCount; i++)
		  sum += trainingData[featureCount * i + tid];
	  
	  return sum / trainingDataCount;
  }
};


//
//Calculates the standard deviation for a column major ordered matrix
//
struct STDFunctor : public thrust::binary_function<int, float, float>
{
	float * trainingData;
	unsigned int trainingDataCount;
	unsigned int featureCount;

	STDFunctor(float * _trainingData, unsigned int _trainingDataCount, unsigned int _featureCount ) : trainingData(_trainingData), trainingDataCount(_trainingDataCount), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float meanValue)
  {
	  float sum = 0;
	  for(int i = 0; i < trainingDataCount; i++)
		  sum += powf(trainingData[featureCount * i + tid] - meanValue, 2.0);
	  
	  return sqrtf(sum / (float)trainingDataCount);
  }
};

//
//Runs the first part of the training
//
struct TrainFunctor : public thrust::binary_function<int, float, float>
{
	float * trainingData;
	float * hypothesis;
	unsigned int featureCount;

	TrainFunctor(float * _trainingData, float * _hypothesis, int _featureCount) : trainingData(_trainingData), hypothesis(_hypothesis), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float labelData)
  {
	  float res=0;
	  for (int i=0;i<featureCount;i++)
		  res+=hypothesis[i]*trainingData[tid*featureCount+i];
	  res-=labelData;
	  return res;
  }
};

//
//Runs the first part of the training
//
struct TrainFunctor2 : public thrust::unary_function<int, float>
{
	float * trainingData;
	float * costData;
	unsigned int featureNumber;
	unsigned int featureCount;

	TrainFunctor2(float * _costData, float * _trainingData, unsigned int _featureCount) : costData(_costData), trainingData(_trainingData), featureCount(_featureCount), featureNumber(0)
	{}

	void SetFeatureNumber(unsigned int value)
	{
		featureNumber = value;
	}

  __host__ __device__
  float operator()(int tid)
  {
	  //if (trainingData[tid * featureCount + featureNumber]==0)
		 // return 0;
	  return costData[tid] * trainingData[tid * featureCount + featureNumber];
  }
};
//
//Applies feature normalization algorithm to the data. (data - mean) / standard deviation
//
struct FeatureNormalizationgFunctor : public thrust::binary_function<int, float, float>
{
	float * meanValue;
	float * stdValue;
	unsigned int featureCount;

	FeatureNormalizationgFunctor(float * _meanValue, float * _stdValue, unsigned int _featureCount) : meanValue(_meanValue), stdValue(_stdValue), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float trainingData)
  {
	  if (stdValue[tid%featureCount]==0)
		  return 1;
	  return (trainingData-meanValue[tid%featureCount])/stdValue[tid%featureCount];
  }
};

//
//Applies the hypothesis to the test data
//
struct PredictFunctor : public thrust::unary_function<int, float>
{
	float * testData;
	float * hypothesis;
	unsigned int featureCount;

	PredictFunctor(float * _testData, float * _hypothesis, unsigned int _featureCount) : testData(_testData), hypothesis(_hypothesis), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid)
  {
	  float price;
	  for (int i=0;i<featureCount;i++)
		  price+=hypothesis[i]*testData[tid*featureCount+i];
	  return price;
  }
};